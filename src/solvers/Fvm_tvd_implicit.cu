#include "hip/hip_runtime.h"
/*
 * Fvm_tvd_implicit.cpp
 *
 *  Created on: Oct 11, 2019
 *      Author: v1
 */

#include "Fvm_tvd_implicit.h"
#include "../tinyxml/tinyxml.h"
#include "../mesh_properties/CellFluidDynamicsProps.h"

#include <string.h>
#include <vector>
#include <algorithm>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <unistd.h>

#include <ctime>

#define INLET_TYPE 4
#define OUTLET_TYPE 1
#define WALL_TYPE 2


#define TETRA_FACES_CNT 4
#define INLET_BND_SIZE 50

__constant__ float* TAU_dev;
__constant__ float inlet_ro_dev[INLET_BND_SIZE];
__constant__ float inlet_ru_dev[INLET_BND_SIZE];
__constant__ float inlet_rv_dev[INLET_BND_SIZE];
__constant__ float inlet_rw_dev[INLET_BND_SIZE];
__constant__ float inlet_rE_dev[INLET_BND_SIZE];
__constant__ float inlet_P_dev[INLET_BND_SIZE];
__constant__ float inlet_gamma_dev[INLET_BND_SIZE];



double FVM_TVD_IMPLICIT::_max(double x, double y)
{
	return (x > y) ? x : y;
}


FVM_TVD_IMPLICIT::FVM_TVD_IMPLICIT()
{
	grid = 0;

	Flux = new double[5];
	Flux1 = new double[5];
	Flux2 = new double[5];
}

FVM_TVD_IMPLICIT::~FVM_TVD_IMPLICIT()
{
	//cout << "FVM_TVD_IMPLICIT" << endl;
	delete [] Flux;
	delete [] Flux1;
	delete [] Flux2;

	if(grid != 0)
	{
		delete grid;
	}
}



void FVM_TVD_IMPLICIT::init(char* xmlFileName)
{
	TiXmlDocument doc( xmlFileName );
	bool loadOkay = doc.LoadFile( TIXML_ENCODING_UTF8 );
	if (!loadOkay)
	{
		Logger::Instance()->logging()->error("Failed to open file : \"%s\"", xmlFileName);
		Logger::Instance()->EXIT(doc.ErrorId());
	}

	double ro, u, v, w, P, gamma;

	TiXmlNode* task = 0;
	TiXmlElement* el = 0;
	TiXmlNode* node0 = 0;
	TiXmlNode* node1 = 0;
	task = doc.FirstChild( "task" );


	node0 = task->FirstChild("mesh");
	const char* fileType = task->FirstChild("mesh")->FirstChild("fileType")->ToElement()->Attribute("value");
	const char* fName = task->FirstChild("mesh")->FirstChild("name")->ToElement()->Attribute("value");

	if(fileType == 0)
	{
		Logger::Instance()->logging()->error("Filetype of Mesh error");
		Logger::Instance()->EXIT(-1);
	}

	if(fName == 0)
	{
		Logger::Instance()->logging()->error("Filename of Mesh error");
		Logger::Instance()->EXIT(-1);
	}

	grid = new Grid(fileType);
	grid->read(fName);

	msh = grid->get_mesh();

	int steadyVal = 1;
	node0 = task->FirstChild("control");
	//node0->FirstChild("STEADY")->ToElement()->Attribute("value", &steadyVal);
	node0->FirstChild("TAU")->ToElement()->Attribute("value", &TAU);
	node0->FirstChild("TMAX")->ToElement()->Attribute("value", &TMAX);
	node0->FirstChild("STEP_MAX")->ToElement()->Attribute("value", &STEP_MAX);
	node0->FirstChild("FILE_OUTPUT_STEP")->ToElement()->Attribute("value", &FILE_STEP_SAVE);
	node0->FirstChild("LOG_OUTPUT_STEP")->ToElement()->Attribute("value", &LOG_STEP_SAVE);

	/*
	const char * flxStr = node0->FirstChild("FLUX")->ToElement()->Attribute("value");
	if (strcmp(flxStr, "GODUNOV") == 0) {
		FLUX = FLUX_GODUNOV;
	}
	else if (strcmp(flxStr, "LAX") == 0) {
		FLUX = FLUX_LAX;
	}
	else {
		FLUX = FLUX_GODUNOV;
	}

	if (steadyVal == 0) {
		STEADY = false;
	} else {
		STEADY = true;
		node1 = node0->FirstChild("CFL");
		node1->FirstChild("start")->ToElement()->Attribute("value", &CFL);
		node1->FirstChild("scale")->ToElement()->Attribute("value", &scaleCFL);
		node1->FirstChild("max")->ToElement()->Attribute("value", &maxCFL);
		node1->FirstChild("step")->ToElement()->Attribute("value", &stepCFL);
		node1->FirstChild("max_limited_cells")->ToElement()->Attribute("value", &maxLimCells);
	}


	int smUsing = 1;
	node0 = task->FirstChild("smoothing");
	node0->FirstChild("using")->ToElement()->Attribute("value", &smUsing);
	node0->FirstChild("coefficient")->ToElement()->Attribute("value", &SMOOTHING_PAR);
	SMOOTHING = (smUsing == 1);


	node0 = task->FirstChild("limits");
	node0->FirstChild("ro")->ToElement()->Attribute("min", &limitRmin);
	node0->FirstChild("ro")->ToElement()->Attribute("max", &limitRmax);
	node0->FirstChild("p")->ToElement()->Attribute( "min", &limitPmin);
	node0->FirstChild("p")->ToElement()->Attribute( "max", &limitPmax);
	node0->FirstChild("u")->ToElement()->Attribute( "max", &limitUmax);


	node0 = task->FirstChild("materials");
	node0->ToElement()->Attribute("count", &matCount);;
	materials = new Material[matCount];
	TiXmlNode* matNode = node0->FirstChild("material");
	for (int i = 0; i < matCount; i++)
	{
		Material & mat = materials[i];
		matNode->ToElement()->Attribute("id", &mat.id);
		node1 = matNode->FirstChild("name");
		el = node1->ToElement();
		mat.name = el->GetText();
		node1 = matNode->FirstChild("parameters");
		node1->FirstChild( "M"  )->ToElement()->Attribute( "value", &mat.M  );
		node1->FirstChild( "Cp" )->ToElement()->Attribute( "value", &mat.Cp );
		node1->FirstChild( "K"  )->ToElement()->Attribute( "value", &mat.K  );
		node1->FirstChild( "ML" )->ToElement()->Attribute( "value", &mat.ML );
		matNode = matNode->NextSibling("material");
	}
	*/

	node0 = task->FirstChild("regions");
	int regCount;
	node0->ToElement()->Attribute("count", &regCount);

	TiXmlNode* regNode = node0->FirstChild("region");
	for (int i = 0; i < regCount; i++)
	{
		node1 = regNode->FirstChild("parameters");

		node1->FirstChild( "ro" )->ToElement()->Attribute( "value", &ro );
		node1->FirstChild( "Vx" )->ToElement()->Attribute( "value", &u );
		node1->FirstChild( "Vy" )->ToElement()->Attribute( "value", &v );
		node1->FirstChild( "Vz" )->ToElement()->Attribute( "value", &w );
		node1->FirstChild( "P"  )->ToElement()->Attribute( "value", &P );
		node1->FirstChild( "Gamma"  )->ToElement()->Attribute( "value", &gamma );

		for (Mesh::CellIterator it = msh->beginCell(), ite = msh->endCell(); it != ite; ++it)
		{
			it->cellFDP.ro = ro;
			it->cellFDP.ru = u * ro;
			it->cellFDP.rv = v * ro;
			it->cellFDP.rw = w * ro;
			it->cellFDP.gamma = gamma;
			it->cellFDP.P = P;

			it->cellFDP.rE = CellFluidDynamicsProps::calc_rE(ro, P, u, v, w, gamma);
		}

		regNode = regNode->NextSibling("region");
	}


	node0 = task->FirstChild("boundaries");
	double bCount;
	node0->ToElement()->Attribute("count", &bCount);
	TiXmlNode* bNode = node0->FirstChild("boundCond");

	for (int i = 0; i < bCount; i++)
	{
		const char * name = bNode->FirstChild("name")->ToElement()->GetText();
		const char * str = bNode->FirstChild("type")->ToElement()->GetText();

		if (strcmp(str, "BOUND_WALL") == 0)
		{
			bndWallNames.push_back(name);
			for (Mesh::FaceIterator it = msh->beginBndFace(name), ite = msh->endBndFace(name); it != ite; ++it)
			{
				it->bnd_type = Face::BND_TYPE_WALL;
			}
		}
		else if (strcmp(str, "BOUND_OUTLET") == 0)
		{
			bndOutletNames.push_back(name);
			for (Mesh::FaceIterator it = msh->beginBndFace(name), ite = msh->endBndFace(name); it != ite; ++it)
			{
				it->bnd_type = Face::BND_TYPE_OUTLET;
			}
		}
		else if (strcmp(str, "BOUND_INLET") == 0)
		{
			bndInletNames.push_back(name);

			node1 = bNode->FirstChild("parameters");

			node1->FirstChild( "ro" )->ToElement()->Attribute( "value", &ro );
			node1->FirstChild( "Vx" )->ToElement()->Attribute( "value", &u );
			node1->FirstChild( "Vy" )->ToElement()->Attribute( "value", &v );
			node1->FirstChild( "Vz" )->ToElement()->Attribute( "value", &w );
			node1->FirstChild( "P"  )->ToElement()->Attribute( "value", &P );
			node1->FirstChild( "Gamma"  )->ToElement()->Attribute( "value", &gamma );

			for (Mesh::FaceIterator it = msh->beginBndFace(name), ite = msh->endBndFace(name); it != ite; ++it)
			{
				it->bnd_type = Face::BND_TYPE_INLET;
				it->faceFDP.ro = ro;
				it->faceFDP.ru = u * ro;
				it->faceFDP.rv = v * ro;
				it->faceFDP.rw = w * ro;
				it->faceFDP.gamma = gamma;
				it->faceFDP.P = P;

				it->faceFDP.rE = CellFluidDynamicsProps::calc_rE(ro, P, u, v, w, gamma);
			}
		}
		else
		{
			Logger::Instance()->logging()->error("Unsupported boundary condition type \"%s\"", str);
			Logger::Instance()->EXIT(1);
		}

		bNode = bNode->NextSibling("boundCond");
	}

	bool check = check_bnd_cond();

	if(!check)
	{
		Logger::Instance()->logging()->error("Boundary names from \"%s\" != boundary names from \"%s\"", xmlFileName, fName);
		Logger::Instance()->EXIT(1);
	}

	save(0);
}

int FVM_TVD_IMPLICIT::check_bnd_cond()
{
	vector<string> v1, v2;

	for(int i = 0; i < bndInletNames.size(); i++)
	{
		v1.push_back(bndInletNames[i]);
	}

	for(int i = 0; i < bndOutletNames.size(); i++)
	{
		v1.push_back(bndOutletNames[i]);
	}

	for(int i = 0; i < bndWallNames.size(); i++)
	{
		v1.push_back(bndWallNames[i]);
	}

	for( map<string, vector<Face*> >::iterator it = msh->bnd_faces.begin(); it != msh->bnd_faces.end(); ++it)
	{
		v2.push_back(it->first);
	}

	sort(v1.begin(), v1.end());
	sort(v2.begin(), v2.end());

	return ( v1.size() == v2.size() && std::equal(v1.begin(), v1.end(), v2.begin()) );
}


void FVM_TVD_IMPLICIT::done()
{
	//free_mem(temp_mat);

	//delete [] Flux;
	//delete [] Flux1;
	//delete [] Flux2;
}


void FVM_TVD_IMPLICIT::flux_Lax_Friedrichs(double* Flux, const CellFluidDynamicsProps& cfdp1, const CellFluidDynamicsProps& cfdp2, const Point& n)
{
	double v_n1 = (cfdp1.ru * n.x + cfdp1.rv * n.y + cfdp1.rw * n.z) / cfdp1.ro;
	double v_n2 = (cfdp2.ru * n.x + cfdp2.rv * n.y + cfdp2.rw * n.z) / cfdp2.ro;

	Flux1[0] = cfdp1.ro * v_n1;
	Flux1[1] = cfdp1.ru * v_n1 + cfdp1.P * n.x;
	Flux1[2] = cfdp1.rv * v_n1 + cfdp1.P * n.y;
	Flux1[3] = cfdp1.rw * v_n1 + cfdp1.P * n.z;
	Flux1[4] = ( cfdp1.rE + cfdp1.P ) * v_n1;

	Flux2[0] = cfdp2.ro * v_n2;
	Flux2[1] = cfdp2.ru * v_n2 + cfdp2.P * n.x;
	Flux2[2] = cfdp2.rv * v_n2 + cfdp2.P * n.y;
	Flux2[3] = cfdp2.rw * v_n2 + cfdp2.P * n.z;
	Flux2[4] = ( cfdp2.rE + cfdp2.P ) * v_n2;

	double eigen_val1 = sqrt(cfdp1.gamma * cfdp1.P / cfdp1.ro) + abs( v_n1 );
	double eigen_val2 = sqrt(cfdp2.gamma * cfdp2.P / cfdp2.ro) + abs( v_n2 );
	double alpha = _max(eigen_val1, eigen_val2);

	Flux[0] = 0.5 * ( Flux1[0] + Flux2[0] - alpha * (cfdp2.ro - cfdp1.ro) );
	Flux[1] = 0.5 * ( Flux1[1] + Flux2[1] - alpha * (cfdp2.ru - cfdp1.ru) );
	Flux[2] = 0.5 * ( Flux1[2] + Flux2[2] - alpha * (cfdp2.rv - cfdp1.rv) );
	Flux[3] = 0.5 * ( Flux1[3] + Flux2[3] - alpha * (cfdp2.rw - cfdp1.rw) );
	Flux[4] = 0.5 * ( Flux1[4] + Flux2[4] - alpha * (cfdp2.rE - cfdp1.rE) );
}


#define TAUU 1E-5


__global__ void calc_fluxes(float* ro, float* ru, float* rv, float* rw, float* rE, float* P, float* gamma, float* n_x, float* n_y, float* n_z, float* S, int* inds_cell, float* fluxes, int nc, float t)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;

	//printf("%d \n", tid);

	float flux1[5];
	float flux2[5];

	while(tid < nc)
	{
		fluxes[5 * tid + 0] = 0;
		fluxes[5 * tid + 1] = 0;
		fluxes[5 * tid + 2] = 0;
		fluxes[5 * tid + 3] = 0;
		fluxes[5 * tid + 4] = 0;

		for(int i = 0; i < TETRA_FACES_CNT; i++)
		{
			int ind = inds_cell[TETRA_FACES_CNT * tid + i];

			if( ind >= 0 )
			{
				float v_n = (ru[tid] * n_x[TETRA_FACES_CNT * tid + i] + rv[tid] * n_x[TETRA_FACES_CNT * tid + i] + rw[tid] * n_z[TETRA_FACES_CNT * tid + i]) / ro[tid];
				float v_n_i = (ru[ind] * n_x[TETRA_FACES_CNT * tid + i] + rv[ind] * n_x[TETRA_FACES_CNT * tid + i] + rw[ind] * n_z[TETRA_FACES_CNT * tid + i]) / ro[ind];

				flux1[0] = ro[tid] * v_n;
				flux1[1] = ru[tid] * v_n + P[tid] * n_x[TETRA_FACES_CNT * tid + i];
				flux1[2] = rv[tid] * v_n + P[tid] * n_y[TETRA_FACES_CNT * tid + i];
				flux1[3] = rw[tid] * v_n + P[tid] * n_z[TETRA_FACES_CNT * tid + i];
				flux1[4] = ( rE[tid] + P[tid] ) * v_n;

				flux2[0] = ro[ind] * v_n_i;
				flux2[1] = ru[ind] * v_n_i + P[ind] * n_x[TETRA_FACES_CNT * tid + i];
				flux2[2] = rv[ind] * v_n_i + P[ind] * n_y[TETRA_FACES_CNT * tid + i];
				flux2[3] = rw[ind] * v_n_i + P[ind] * n_z[TETRA_FACES_CNT * tid + i];
				flux2[4] = ( rE[ind] + P[ind] ) * v_n_i;

				float eigen_val1 = sqrtf(gamma[tid] * P[tid] / ro[tid]) + fabsf( v_n );
				float eigen_val2 = sqrtf(gamma[ind] * P[ind] / ro[ind]) + fabsf( v_n_i );
				float alpha = fmaxf(eigen_val1, eigen_val2);

				fluxes[5 * tid + 0] = fluxes[5 * tid + 0] + 0.5 * ( flux1[0] + flux2[0] - alpha * (ro[ind] - ro[tid]) ) * S[TETRA_FACES_CNT * tid + i];
				fluxes[5 * tid + 1] = fluxes[5 * tid + 1] + 0.5 * ( flux1[1] + flux2[1] - alpha * (ru[ind] - ru[tid]) ) * S[TETRA_FACES_CNT * tid + i];
				fluxes[5 * tid + 2] = fluxes[5 * tid + 2] + 0.5 * ( flux1[2] + flux2[2] - alpha * (rv[ind] - rv[tid]) ) * S[TETRA_FACES_CNT * tid + i];
				fluxes[5 * tid + 3] = fluxes[5 * tid + 3] + 0.5 * ( flux1[3] + flux2[3] - alpha * (rw[ind] - rw[tid]) ) * S[TETRA_FACES_CNT * tid + i];
				fluxes[5 * tid + 4] = fluxes[5 * tid + 4] + 0.5 * ( flux1[4] + flux2[4] - alpha * (rE[ind] - rE[tid]) ) * S[TETRA_FACES_CNT * tid + i];
			}
			else
			{
				if( ind == - OUTLET_TYPE )
				{
					float v_n = (ru[tid] * n_x[TETRA_FACES_CNT * tid + i] + rv[tid] * n_x[TETRA_FACES_CNT * tid + i] + rw[tid] * n_z[TETRA_FACES_CNT * tid + i]) / ro[tid];

					flux1[0] = ro[tid] * v_n;
					flux1[1] = ru[tid] * v_n + P[tid] * n_x[TETRA_FACES_CNT * tid + i];
					flux1[2] = rv[tid] * v_n + P[tid] * n_y[TETRA_FACES_CNT * tid + i];
					flux1[3] = rw[tid] * v_n + P[tid] * n_z[TETRA_FACES_CNT * tid + i];
					flux1[4] = ( rE[tid] + P[tid] ) * v_n;

					fluxes[5 * tid + 0] = fluxes[5 * tid + 0] + flux1[0] * S[TETRA_FACES_CNT * tid + i];
					fluxes[5 * tid + 1] = fluxes[5 * tid + 1] + flux1[1] * S[TETRA_FACES_CNT * tid + i];
					fluxes[5 * tid + 2] = fluxes[5 * tid + 2] + flux1[2] * S[TETRA_FACES_CNT * tid + i];
					fluxes[5 * tid + 3] = fluxes[5 * tid + 3] + flux1[3] * S[TETRA_FACES_CNT * tid + i];
					fluxes[5 * tid + 4] = fluxes[5 * tid + 4] + flux1[4] * S[TETRA_FACES_CNT * tid + i];
				}
				else if( ind == - WALL_TYPE )
				{
					float v_n = (ru[tid] * n_x[TETRA_FACES_CNT * tid + i] + rv[tid] * n_x[TETRA_FACES_CNT * tid + i] + rw[tid] * n_z[TETRA_FACES_CNT * tid + i]) / ro[tid];

					float ru2 = ru[tid] - 2 * v_n * ro[tid] * n_x[TETRA_FACES_CNT * tid + i];
					float rv2 = rv[tid] - 2 * v_n * ro[tid] * n_y[TETRA_FACES_CNT * tid + i];
					float rw2 = rw[tid] - 2 * v_n * ro[tid] * n_z[TETRA_FACES_CNT * tid + i];
					float rE2 =	P[tid] / (gamma[tid] - 1) + 0.5 * (ru2*ru2 + rv2*rv2 + rw2*rw2) / ro[tid];

					float v_n_i = (ru2 * n_x[TETRA_FACES_CNT * tid + i] + rv2 * n_x[TETRA_FACES_CNT * tid + i] + rw2 * n_z[TETRA_FACES_CNT * tid + i]) / ro[tid];

					flux1[0] = ro[tid] * v_n;
					flux1[1] = ru[tid] * v_n + P[tid] * n_x[TETRA_FACES_CNT * tid + i];
					flux1[2] = rv[tid] * v_n + P[tid] * n_y[TETRA_FACES_CNT * tid + i];
					flux1[3] = rw[tid] * v_n + P[tid] * n_z[TETRA_FACES_CNT * tid + i];
					flux1[4] = ( rE[tid] + P[tid] ) * v_n;

					flux2[0] = ro[tid] * v_n_i;
					flux2[1] = ru2 * v_n_i + P[tid] * n_x[TETRA_FACES_CNT * tid + i];
					flux2[2] = rv2 * v_n_i + P[tid] * n_y[TETRA_FACES_CNT * tid + i];
					flux2[3] = rw2 * v_n_i + P[tid] * n_z[TETRA_FACES_CNT * tid + i];
					flux2[4] = ( rE2 + P[tid] ) * v_n_i;

					float alpha = sqrtf(gamma[tid] * P[tid] / ro[tid]) + fmaxf(fabsf( v_n ), fabsf( v_n_i ));

					fluxes[5 * tid + 0] = fluxes[5 * tid + 0] + 0.5 * ( flux1[0] + flux2[0] ) * S[TETRA_FACES_CNT * tid + i];
					fluxes[5 * tid + 1] = fluxes[5 * tid + 1] + 0.5 * ( flux1[1] + flux2[1] - alpha * (ru2 - ru[tid]) ) * S[TETRA_FACES_CNT * tid + i];
					fluxes[5 * tid + 2] = fluxes[5 * tid + 2] + 0.5 * ( flux1[2] + flux2[2] - alpha * (rv2 - rv[tid]) ) * S[TETRA_FACES_CNT * tid + i];
					fluxes[5 * tid + 3] = fluxes[5 * tid + 3] + 0.5 * ( flux1[3] + flux2[3] - alpha * (rw2 - rw[tid]) ) * S[TETRA_FACES_CNT * tid + i];
					fluxes[5 * tid + 4] = fluxes[5 * tid + 4] + 0.5 * ( flux1[4] + flux2[4] - alpha * (rE2 - rE[tid]) ) * S[TETRA_FACES_CNT * tid + i];
				}
				else
				{
					int inlet_ind = -ind - INLET_TYPE; // -4 -> 0, -5 -> 1, -6 -> 2 ...
					/*
					float ro2 = inlet_ro_dev[inlet_ind];
					float ru2 = inlet_ru_dev[inlet_ind];
					float rv2 = inlet_rv_dev[inlet_ind];
					float rw2 = inlet_rw_dev[inlet_ind];
					float rE2 = inlet_rE_dev[inlet_ind];
					float P2 = inlet_P_dev[inlet_ind];
					float gamma2 = inlet_gamma_dev[inlet_ind];
					*/
					/*
					float ro2 = 0.43567;
					float ru2 = ro2 * 219.521;
					float rv2 = ro2 * 8.85526;
					float rw2 = ro2 * 0;
					float P2 = 28263.7;
					float gamma2 = 1.4;
					float rE2 = P2 / (gamma2 - 1) + 0.5 * (ru2*ru2 + rv2*rv2 + rw2*rw2) / ro2;
					*/

					float ro2 = 1.4;
					float ru2 = ro2 * 0;
					float rv2 = ro2 * 0;
					float rw2 = ro2 * 3;
					float P2 = 1;
					float gamma2 = 1.4;
					float rE2 = P2 / (gamma2 - 1) + 0.5 * (ru2*ru2 + rv2*rv2 + rw2*rw2) / ro2;


					float v_n = (ru[tid] * n_x[TETRA_FACES_CNT * tid + i] + rv[tid] * n_x[TETRA_FACES_CNT * tid + i] + rw[tid] * n_z[TETRA_FACES_CNT * tid + i]) / ro[tid];
					float v_n_i = (ru2 * n_x[TETRA_FACES_CNT * tid + i] + rv2 * n_x[TETRA_FACES_CNT * tid + i] + rw2 * n_z[TETRA_FACES_CNT * tid + i]) / ro2;

					flux1[0] = ro[tid] * v_n;
					flux1[1] = ru[tid] * v_n + P[tid] * n_x[TETRA_FACES_CNT * tid + i];
					flux1[2] = rv[tid] * v_n + P[tid] * n_y[TETRA_FACES_CNT * tid + i];
					flux1[3] = rw[tid] * v_n + P[tid] * n_z[TETRA_FACES_CNT * tid + i];
					flux1[4] = ( rE[tid] + P[tid] ) * v_n;

					flux2[0] = ro2 * v_n_i;
					flux2[1] = ru2 * v_n_i + P2 * n_x[TETRA_FACES_CNT * tid + i];
					flux2[2] = rv2 * v_n_i + P2 * n_y[TETRA_FACES_CNT * tid + i];
					flux2[3] = rw2 * v_n_i + P2 * n_z[TETRA_FACES_CNT * tid + i];
					flux2[4] = ( rE2 + P2 ) * v_n_i;

					float eigen_val1 = sqrtf(gamma[tid] * P[tid] / ro[tid]) + fabsf( v_n );
					float eigen_val2 = sqrtf(gamma2 * P2 / ro2) + fabsf( v_n_i );
					float alpha = fmaxf(eigen_val1, eigen_val2);

					fluxes[5 * tid + 0] = fluxes[5 * tid + 0] + 0.5 * ( flux1[0] + flux2[0] - alpha * (ro2 - ro[tid]) ) * S[TETRA_FACES_CNT * tid + i];
					fluxes[5 * tid + 1] = fluxes[5 * tid + 1] + 0.5 * ( flux1[1] + flux2[1] - alpha * (ru2 - ru[tid]) ) * S[TETRA_FACES_CNT * tid + i];
					fluxes[5 * tid + 2] = fluxes[5 * tid + 2] + 0.5 * ( flux1[2] + flux2[2] - alpha * (rv2 - rv[tid]) ) * S[TETRA_FACES_CNT * tid + i];
					fluxes[5 * tid + 3] = fluxes[5 * tid + 3] + 0.5 * ( flux1[3] + flux2[3] - alpha * (rw2 - rw[tid]) ) * S[TETRA_FACES_CNT * tid + i];
					fluxes[5 * tid + 4] = fluxes[5 * tid + 4] + 0.5 * ( flux1[4] + flux2[4] - alpha * (rE2 - rE[tid]) ) * S[TETRA_FACES_CNT * tid + i];
				}
			}
		}

		//printf("%f %f %f %f %f %f\n",fluxes[5 * tid + 0], fluxes[5 * tid + 1], fluxes[5 * tid + 2], fluxes[5 * tid + 3], fluxes[5 * tid + 4], t);

		tid += blockDim.x * gridDim.x;
	}
}

__device__ float calc_P_dev(float ro, float rE, float ru, float rv, float rw, float gamma)
{
	return (rE - 0.5 *(ru*ru + rv*rv + rw*rw) / ro) * (gamma - 1);
}


__global__ void calc_new_values(float* ro, float* ru, float* rv, float* rw, float* rE, float* P, float* gamma, float* V, float* fluxes, int nc)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;

	while(tid < nc)
	{
		ro[tid] = ro[tid] - TAUU * fluxes[5 * tid + 0] / V[tid];
		ru[tid] = ru[tid] - TAUU * fluxes[5 * tid + 1] / V[tid];
		rv[tid] = rv[tid] - TAUU * fluxes[5 * tid + 2] / V[tid];
		rw[tid] = rw[tid] - TAUU * fluxes[5 * tid + 3] / V[tid];
		rE[tid] = rE[tid] - TAUU * fluxes[5 * tid + 4] / V[tid];

		P[tid] = calc_P_dev(ro[tid], rE[tid], ru[tid], rv[tid], rw[tid], gamma[tid]);
		//printf("%f %f %f %f %f %f\n",fluxes[5 * tid + 0], fluxes[5 * tid + 1], fluxes[5 * tid + 2], fluxes[5 * tid + 3], fluxes[5 * tid + 4], P[tid]);
		tid += blockDim.x * gridDim.x;
	}
}

void FVM_TVD_IMPLICIT::run()
{
	Logger::Instance()->logging()->info("TMAX = %e STEP_MAX = %d", TMAX, STEP_MAX);

	unsigned int nc = msh->cells.size();

	double t = 0;
	int step = 0;

	int num_bytes_cells = nc * sizeof(float);

	float* buffer = new float[ nc ];

	float* V_dev = 0;
	float* fluxes_dev = 0;
	float* ro_dev = 0;
	float* ru_dev = 0;
	float* rv_dev = 0;
	float* rw_dev = 0;
	float* rE_dev = 0;
	float* P_dev = 0;
	float* gamma_dev = 0;


	hipMalloc( (void**)&V_dev, num_bytes_cells );
	hipMalloc( (void**)&fluxes_dev, 5 * nc * sizeof(float) );
    hipMalloc( (void**)&ro_dev, num_bytes_cells );
    hipMalloc( (void**)&ru_dev, num_bytes_cells );
    hipMalloc( (void**)&rv_dev, num_bytes_cells );
    hipMalloc( (void**)&rw_dev, num_bytes_cells );
    hipMalloc( (void**)&rE_dev, num_bytes_cells );
    hipMalloc( (void**)&P_dev, num_bytes_cells );
    hipMalloc( (void**)&gamma_dev, num_bytes_cells );


    hipMemcpyToSymbol(HIP_SYMBOL(TAU_dev), &TAU, sizeof(float), hipMemcpyHostToDevice);

    for (Mesh::CellIterator it = msh->beginCell(), ite = msh->endCell(); it != ite; ++it)
	{
		buffer[it->index] = it->V;
	}
	hipMemcpy(V_dev, buffer, num_bytes_cells, hipMemcpyHostToDevice);


	////////////////////////

    for (Mesh::CellIterator it = msh->beginCell(), ite = msh->endCell(); it != ite; ++it)
	{
		buffer[it->index] = it->cellFDP.ro;
	}
    hipMemcpy(ro_dev, buffer, num_bytes_cells, hipMemcpyHostToDevice);


    for (Mesh::CellIterator it = msh->beginCell(), ite = msh->endCell(); it != ite; ++it)
	{
		buffer[it->index] = it->cellFDP.ru;
	}
    hipMemcpy(ru_dev, buffer, num_bytes_cells, hipMemcpyHostToDevice);


    for (Mesh::CellIterator it = msh->beginCell(), ite = msh->endCell(); it != ite; ++it)
	{
		buffer[it->index] = it->cellFDP.rv;
	}
	hipMemcpy(rv_dev, buffer, num_bytes_cells, hipMemcpyHostToDevice);


	for (Mesh::CellIterator it = msh->beginCell(), ite = msh->endCell(); it != ite; ++it)
	{
		buffer[it->index] = it->cellFDP.rw;
	}
	hipMemcpy(rw_dev, buffer, num_bytes_cells, hipMemcpyHostToDevice);

	for (Mesh::CellIterator it = msh->beginCell(), ite = msh->endCell(); it != ite; ++it)
	{
		buffer[it->index] = it->cellFDP.rE;
	}
	hipMemcpy(rE_dev, buffer, num_bytes_cells, hipMemcpyHostToDevice);

	for (Mesh::CellIterator it = msh->beginCell(), ite = msh->endCell(); it != ite; ++it)
	{
		buffer[it->index] = it->cellFDP.P;
	}
	hipMemcpy(P_dev, buffer, num_bytes_cells, hipMemcpyHostToDevice);


	for (Mesh::CellIterator it = msh->beginCell(), ite = msh->endCell(); it != ite; ++it)
	{
		buffer[it->index] = it->cellFDP.gamma;
	}
	hipMemcpy(gamma_dev, buffer, num_bytes_cells, hipMemcpyHostToDevice);


	//////////////////


	int num_bytes_faces = TETRA_FACES_CNT * nc * sizeof(float);

	int* buffer_ind = new int[TETRA_FACES_CNT * nc];
	float* buffer_faces = new float[TETRA_FACES_CNT * nc];

	float* n_x_dev = 0;
	float* n_y_dev = 0;
	float* n_z_dev = 0;
	float* S_dev = 0;

	int* inds_cell_dev = 0;

	hipMalloc( (void**)&n_x_dev, num_bytes_faces );
	hipMalloc( (void**)&n_y_dev, num_bytes_faces );
	hipMalloc( (void**)&n_z_dev, num_bytes_faces );
	hipMalloc( (void**)&S_dev, num_bytes_faces );
	hipMalloc( (void**)&inds_cell_dev, TETRA_FACES_CNT * nc * sizeof(int) );

	for (Mesh::CellIterator it = msh->beginCell(), ite = msh->endCell(); it != ite; ++it)
	{
		for(int i = 0; i < TETRA_FACES_CNT; i++)
		{
			int ic = it->f[i]->in_cell;

			if(it->f[i]->c[ic] == msh->cells[it->index])
			{
				buffer_faces[TETRA_FACES_CNT * it->index + i] = -it->f[i]->n.x;
			}
			else
			{
				buffer_faces[TETRA_FACES_CNT * it->index + i] = it->f[i]->n.x;
			}
		}
	}
	hipMemcpy(n_x_dev, buffer_faces, num_bytes_faces, hipMemcpyHostToDevice);

	for (Mesh::CellIterator it = msh->beginCell(), ite = msh->endCell(); it != ite; ++it)
	{
		for(int i = 0; i < TETRA_FACES_CNT; i++)
		{
			int ic = it->f[i]->in_cell;

			if(it->f[i]->c[ic] == msh->cells[it->index])
			{
				buffer_faces[TETRA_FACES_CNT * it->index + i] = -it->f[i]->n.y;
			}
			else
			{
				buffer_faces[TETRA_FACES_CNT * it->index + i] = it->f[i]->n.y;
			}
		}
	}
	hipMemcpy(n_y_dev, buffer_faces, num_bytes_faces, hipMemcpyHostToDevice);

	for (Mesh::CellIterator it = msh->beginCell(), ite = msh->endCell(); it != ite; ++it)
	{
		for(int i = 0; i < TETRA_FACES_CNT; i++)
		{
			int ic = it->f[i]->in_cell;
			if(it->f[i]->c[ic] == msh->cells[it->index])
			{
				buffer_faces[TETRA_FACES_CNT * it->index + i] = -it->f[i]->n.z;
			}
			else
			{
				buffer_faces[TETRA_FACES_CNT * it->index + i] = it->f[i]->n.z;
			}
		}
	}
	hipMemcpy(n_z_dev, buffer_faces, num_bytes_faces, hipMemcpyHostToDevice);

	for (Mesh::CellIterator it = msh->beginCell(), ite = msh->endCell(); it != ite; ++it)
	{
		for(int i = 0; i < TETRA_FACES_CNT; i++)
		{
			buffer_faces[TETRA_FACES_CNT * it->index + i] = it->f[i]->S;
		}
	}
	hipMemcpy(S_dev, buffer_faces, num_bytes_faces, hipMemcpyHostToDevice);

	for (Mesh::CellIterator it = msh->beginCell(), ite = msh->endCell(); it != ite; ++it)
	{
		for(int i = 0; i < TETRA_FACES_CNT; i++)
		{
			Cell* other_cell = (msh->cells[it->index] == it->f[i]->c[0]) ? it->f[i]->c[1] : it->f[i]->c[0];

			if(other_cell != 0)
			{
				buffer_faces[TETRA_FACES_CNT * it->index + i] = other_cell->index;
			}
			else
			{
				switch(it->f[i]->bnd_type)
				{
					case Face::BND_TYPE_WALL:
						buffer_ind[TETRA_FACES_CNT * it->index + i] = - WALL_TYPE;
						break;

					case Face::BND_TYPE_OUTLET:
						buffer_ind[TETRA_FACES_CNT * it->index + i] = - OUTLET_TYPE;
						break;

					case Face::BND_TYPE_INLET:
					{
						buffer_ind[TETRA_FACES_CNT * it->index + i] = - INLET_TYPE;

						float _ro = it->f[i]->faceFDP.ro;
						float _ru = it->f[i]->faceFDP.ru;
						float _rv = it->f[i]->faceFDP.rv;
						float _rw = it->f[i]->faceFDP.rw;
						float _rE = it->f[i]->faceFDP.rE;
						float _P = it->f[i]->faceFDP.P;
						float _gamma = it->f[i]->faceFDP.gamma;

						hipMemcpyToSymbol(HIP_SYMBOL(inlet_ro_dev), &_ro, sizeof(float), hipMemcpyHostToDevice);
						hipMemcpyToSymbol(HIP_SYMBOL(inlet_ru_dev), &_ru, sizeof(float), hipMemcpyHostToDevice);
						hipMemcpyToSymbol(HIP_SYMBOL(inlet_rv_dev), &_rv, sizeof(float), hipMemcpyHostToDevice);
						hipMemcpyToSymbol(HIP_SYMBOL(inlet_rw_dev), &_rw, sizeof(float), hipMemcpyHostToDevice);
						hipMemcpyToSymbol(HIP_SYMBOL(inlet_rE_dev), &_rE, sizeof(float), hipMemcpyHostToDevice);
						hipMemcpyToSymbol(HIP_SYMBOL(inlet_P_dev), &_P, sizeof(float), hipMemcpyHostToDevice);
						hipMemcpyToSymbol(HIP_SYMBOL(inlet_gamma_dev), &_gamma, sizeof(float), hipMemcpyHostToDevice);

						break;
					}
					default:
						Logger::Instance()->logging()->error("There is boundary face that has not BND_TYPE(WALL, OUTLET, INLET)");
				}
			}
		}
	}
	hipMemcpy(inds_cell_dev, buffer_ind, TETRA_FACES_CNT * nc * sizeof(int), hipMemcpyHostToDevice);

	free(buffer_ind);
	free(buffer_faces);


	Logger::Instance()->logging()->info("complete...");


	dim3 threads = dim3(2, 1, 1);
	//dim3 blocks = dim3(nc / threads.x + threads.x, 1, 1);
	dim3 blocks = dim3(10, 1, 1);

	Logger::Instance()->logging()->info("Solving the equation (FVM_TVD_IMPLICIT)");
	float* _buffer = new float[ 5*nc ];
	while(t < TMAX && step < STEP_MAX)
	{
		long time_start, time_end;
		time_start = clock();

		t += TAU;
		step++;


		calc_fluxes<<<blocks, threads>>> (ro_dev, ru_dev, rv_dev, rw_dev, rE_dev, P_dev, gamma_dev, n_x_dev, n_y_dev, n_z_dev, S_dev, inds_cell_dev, fluxes_dev, nc, t);
		//sleep(1); printf("%f\n",t);

		calc_new_values<<<blocks, threads>>>(ro_dev, ru_dev, rv_dev, rw_dev, rE_dev, P_dev, gamma_dev, V_dev, fluxes_dev, nc);

		hipMemcpy(_buffer, fluxes_dev, 5 * nc * sizeof(float), hipMemcpyDeviceToHost);
		//printf("%f %f %f %f %f\n",_buffer[5 * 4 + 0], _buffer[5 * 4 + 1], _buffer[5 * 4 + 2], _buffer[5 * 4 + 3], _buffer[5 * 4 + 4]);

		/*
		for(Mesh::BndFaceIterator it = msh->beginBndFace(&(msh->bnd_faces), &bndWallNames), ite = msh->endBndFace(&(msh->bnd_faces), &bndWallNames); it != ite; ++it)
		{
			c1 = it->c[0]->index;

            it->faceFDP.ro = it->c[0]->cellFDP.ro;
			it->faceFDP.rE = it->c[0]->cellFDP.rE;
			it->faceFDP.P = it->c[0]->cellFDP.P;
			it->faceFDP.gamma = it->c[0]->cellFDP.gamma;

			double rvel_n = it->c[0]->cellFDP.ru * it->n.x + it->c[0]->cellFDP.rv * it->n.y + it->c[0]->cellFDP.rw * it->n.z;


			it->faceFDP.ru = it->c[0]->cellFDP.ru - 2 * rvel_n * it->n.x;
			it->faceFDP.rv = it->c[0]->cellFDP.rv - 2 * rvel_n * it->n.y;
			it->faceFDP.rw = it->c[0]->cellFDP.rw - 2 * rvel_n * it->n.z;


			flux_Lax_Friedrichs(Flux, it->c[0]->cellFDP, it->faceFDP, it->n);

			for(int i = 0; i < 5; i++)
			{
				right5[ c1 ][i] -= Flux[i] * it->S;

			}

			CellFluidDynamicsProps::calc_Roe_Avg(temp_u, temp_v, temp_w, temp_H, temp_c, temp_GAMMA, it->c[0]->cellFDP, it->faceFDP);

			eigen_values(eigen_vals, temp_u, temp_v, temp_w, temp_c, it->n);
			left_eigen_vecs(left_vecs, temp_u, temp_v, temp_w, temp_c, temp_GAMMA, it->n);
			right_eigen_vecs(right_vecs, temp_u, temp_v, temp_w, temp_c, temp_H, it->n);

			matrix_A(A_plus, right_vecs, eigen_vals, left_vecs, FVM_TVD_IMPLICIT::PLUS_JACOBIAN);

			for(int i = 0; i < 5; i++)
			{
				for(int j = 0; j < 5; j++)
				{
					A_plus[i][j] *= it->S;
				}
			}

			solverMtx->addMatrElement(c1, c1, A_plus);
		}


		for(Mesh::BndFaceIterator it = msh->beginBndFace(&(msh->bnd_faces), &bndInletNames), ite = msh->endBndFace(&(msh->bnd_faces), &bndInletNames); it != ite; ++it)
		{
			c1 = it->c[0]->index;

			flux_Lax_Friedrichs(Flux, it->c[0]->cellFDP, it->faceFDP, it->n);

			for(int i = 0; i < 5; i++)
			{
				right5[ c1 ][i] -= Flux[i] * it->S;

			}

			CellFluidDynamicsProps::calc_Roe_Avg(temp_u, temp_v, temp_w, temp_H, temp_c, temp_GAMMA, it->c[0]->cellFDP, it->faceFDP);

			eigen_values(eigen_vals, temp_u, temp_v, temp_w, temp_c, it->n);
			left_eigen_vecs(left_vecs, temp_u, temp_v, temp_w, temp_c, temp_GAMMA, it->n);
			right_eigen_vecs(right_vecs, temp_u, temp_v, temp_w, temp_c, temp_H, it->n);

			matrix_A(A_plus, right_vecs, eigen_vals, left_vecs, FVM_TVD_IMPLICIT::PLUS_JACOBIAN);

			for(int i = 0; i < 5; i++)
			{
				for(int j = 0; j < 5; j++)
				{
					A_plus[i][j] *= it->S;
				}
			}

			solverMtx->addMatrElement(c1, c1, A_plus);
		}


		for(Mesh::BndFaceIterator it = msh->beginBndFace(&(msh->bnd_faces), &bndOutletNames), ite = msh->endBndFace(&(msh->bnd_faces), &bndOutletNames); it != ite; ++it)
		{
			c1 = it->c[0]->index;

			it->faceFDP.ro = it->c[0]->cellFDP.ro;
			it->faceFDP.ru = it->c[0]->cellFDP.ru;
			it->faceFDP.rv = it->c[0]->cellFDP.rv;
			it->faceFDP.rw = it->c[0]->cellFDP.rw;
			it->faceFDP.rE = it->c[0]->cellFDP.rE;
			it->faceFDP.P = it->c[0]->cellFDP.P;
			it->faceFDP.gamma = it->c[0]->cellFDP.gamma;

			flux_Lax_Friedrichs(Flux, it->c[0]->cellFDP, it->faceFDP, it->n);

			for(int i = 0; i < 5; i++)
			{
				right5[ c1 ][i] -= Flux[i] * it->S;
			}

			CellFluidDynamicsProps::calc_Roe_Avg(temp_u, temp_v, temp_w, temp_H, temp_c, temp_GAMMA, it->c[0]->cellFDP, it->faceFDP);

			eigen_values(eigen_vals, temp_u, temp_v, temp_w, temp_c, it->n);
			left_eigen_vecs(left_vecs, temp_u, temp_v, temp_w, temp_c, temp_GAMMA, it->n);
			right_eigen_vecs(right_vecs, temp_u, temp_v, temp_w, temp_c, temp_H, it->n);

			matrix_A(A_plus, right_vecs, eigen_vals, left_vecs, FVM_TVD_IMPLICIT::PLUS_JACOBIAN);

			for(int i = 0; i < 5; i++)
			{
				for(int j = 0; j < 5; j++)
				{
					A_plus[i][j] *= it->S;
				}
			}

			solverMtx->addMatrElement(c1, c1, A_plus);
		}



		for(Mesh::FaceIterator it = msh->beginInnerFace(), ite = msh->endInnerFace(); it != ite; ++it)
		{
			oc = it->out_cell;
			ic = it->in_cell;
			c1 = it->c[oc]->index;
			c2 = it->c[ic]->index;

			flux_Lax_Friedrichs(Flux, it->c[oc]->cellFDP, it->c[ic]->cellFDP, it->n);

			for(int i = 0; i < 5; i++)
			{
				right5[ c1 ][i] -= Flux[i] * it->S;
				right5[ c2 ][i] += Flux[i] * it->S;
			}

			CellFluidDynamicsProps::calc_Roe_Avg(temp_u, temp_v, temp_w, temp_H, temp_c, temp_GAMMA, it->c[0]->cellFDP, it->c[1]->cellFDP);

			eigen_values(eigen_vals, temp_u, temp_v, temp_w, temp_c, it->n);
			left_eigen_vecs(left_vecs, temp_u, temp_v, temp_w, temp_c, temp_GAMMA, it->n);
			right_eigen_vecs(right_vecs, temp_u, temp_v, temp_w, temp_c, temp_H, it->n);


			matrix_A(A_plus, right_vecs, eigen_vals, left_vecs, FVM_TVD_IMPLICIT::PLUS_JACOBIAN);
			matrix_A(A_minus, right_vecs, eigen_vals, left_vecs, FVM_TVD_IMPLICIT::MINUS_JACOBIAN);


			for(int i = 0; i < 5; i++)
			{
				for(int j = 0; j < 5; j++)
				{
					A_plus[i][j]  *= it->S;
					A_minus[i][j] *= it->S;
				}
			}

			solverMtx->addMatrElement(c1, c1, A_plus);
			solverMtx->addMatrElement(c1, c2, A_minus);

			pc.x = -it->n.x;
			pc.y = -it->n.y;
			pc.z = -it->n.z;

			eigen_values(eigen_vals, temp_u, temp_v, temp_w, temp_c, pc);
			left_eigen_vecs(left_vecs, temp_u, temp_v, temp_w, temp_c, temp_GAMMA, pc);
			right_eigen_vecs(right_vecs, temp_u, temp_v, temp_w, temp_c, temp_H, pc);

			matrix_A(A_plus, right_vecs, eigen_vals, left_vecs, FVM_TVD_IMPLICIT::PLUS_JACOBIAN);
			matrix_A(A_minus, right_vecs, eigen_vals, left_vecs, FVM_TVD_IMPLICIT::MINUS_JACOBIAN);


			for(int i = 0; i < 5; i++)
			{
				for(int j = 0; j < 5; j++)
				{
					A_plus[i][j]  *= it->S;
					A_minus[i][j] *= it->S;
				}
			}

			solverMtx->addMatrElement(c2, c2, A_plus);
			solverMtx->addMatrElement(c2, c1, A_minus);
		}


		for(Mesh::CellIterator it = msh->beginCell(), ite = msh->endCell(); it != ite; ++it)
		{
			c1 = it->index;
			double V_tau = it->V / TAU;

			for(int i = 0; i < 5; i++)
			{
				mtx5[i][i] = V_tau;
			}

			solverMtx->addMatrElement(c1, c1, mtx5);
			solverMtx->setRightElement(c1, right5[c1]);
		}


		solveErr = solverMtx->solve(eps, max_iter);
		*/

		time_end = clock();

		if(step % FILE_STEP_SAVE == 0)
		{
			hipMemcpy(buffer, ro_dev, num_bytes_cells, hipMemcpyDeviceToHost);
			for (Mesh::CellIterator it = msh->beginCell(), ite = msh->endCell(); it != ite; ++it)
			{
				it->cellFDP.ro = buffer[it->index];
			}

			hipMemcpy(buffer, ru_dev, num_bytes_cells, hipMemcpyDeviceToHost);
			for (Mesh::CellIterator it = msh->beginCell(), ite = msh->endCell(); it != ite; ++it)
			{
				it->cellFDP.ru = buffer[it->index];
			}

			hipMemcpy(buffer, rv_dev, num_bytes_cells, hipMemcpyDeviceToHost);
			for (Mesh::CellIterator it = msh->beginCell(), ite = msh->endCell(); it != ite; ++it)
			{
				it->cellFDP.rv = buffer[it->index];
			}

			hipMemcpy(buffer, rw_dev, num_bytes_cells, hipMemcpyDeviceToHost);
			for (Mesh::CellIterator it = msh->beginCell(), ite = msh->endCell(); it != ite; ++it)
			{
				it->cellFDP.rw = buffer[it->index];;
			}

			hipMemcpy(buffer, P_dev, num_bytes_cells, hipMemcpyDeviceToHost);
			for (Mesh::CellIterator it = msh->beginCell(), ite = msh->endCell(); it != ite; ++it)
			{
				it->cellFDP.P = buffer[it->index];
			}

			save(step);
		}

		if(step % LOG_STEP_SAVE == 0)
		{
			Logger::Instance()->logging()->info("step : %d\ttime step : %.16f\ttime: %d ticks", step, t, time_end - time_start);
		}

	}

    save(step);
    Logger::Instance()->logging()->info("complete...");

}




void FVM_TVD_IMPLICIT::save(int step)
{
    FILE *out;
    char c[20];

    sprintf(c, "res_%d.vtk", step);
    out = fopen(c, "w");
    fprintf(out, "# vtk DataFile Version 3.0\n");
    //The header can be used to describe the data
    fprintf(out, "GASDIN data file\n");
    fprintf(out, "ASCII\n");
    fprintf(out, "DATASET UNSTRUCTURED_GRID\n");

    fprintf(out, "POINTS %d double\n", msh->pCount);
    for (int i = 0; i < msh->pCount; i++)
    {
        fprintf(out, "%f %f %f\n", msh->points[i].x, msh->points[i].y, msh->points[i].z);
    }

    int cellCount = msh->cells.size();

    /*
    cellSize + cellCount :
    cellSize + one number for each cell - count of points in this cell
    */
    fprintf(out, "CELLS %d %d\n", cellCount, msh->cnt_of_points + cellCount);
    for (int i = 0; i < cellCount; i++)
    {
        fprintf(out, "%d", msh->cells[i]->pCount);

        for (int k = 0; k < msh->cells[i]->pCount; k++)
        {
            fprintf(out, " %d", msh->cells[i]->p[k]->index);
        }

        fprintf(out, "\n");
    }

    fprintf(out, "CELL_TYPES %d\n", cellCount);
    for (int i = 0; i < cellCount; i++)
    {
        switch (msh->cells[i]->type)
        {
			case Mesh::TYPE_TETRAHEDRON:
			{
				fprintf(out, "10\n"); //10 - VTK_TETRA
				break;
			}
			case Mesh::TYPE_WEDGE:
			{
				fprintf(out, "13\n"); //13 - VTK_WEDGE
				break;
			}
			case Mesh::TYPE_HEXAHEDRON:
			{
				fprintf(out, "12\n"); //12 - VTK_HEXAHEDRON
				break;
			}
        }
    }


	fprintf(out, "CELL_DATA %d\nSCALARS Density double 1\nLOOKUP_TABLE default\n", cellCount);
	for (int i = 0; i < cellCount; i++)
	{
	  fprintf(out, "%25.16f\n", msh->cells[i]->cellFDP.ro);
	}

	fprintf(out, "SCALARS Pressure double 1\nLOOKUP_TABLE default\n");
	for (int i = 0; i < cellCount; i++)
	{
	   fprintf(out, "%25.16f\n", msh->cells[i]->cellFDP.P);
	}

	fprintf(out, "SCALARS Mach double 1\nLOOKUP_TABLE default\n");
	for (int i = 0; i < cellCount; i++)
	{
        double ro = msh->cells[i]->cellFDP.ro;
        double u = msh->cells[i]->cellFDP.ru / ro;
        double v = msh->cells[i]->cellFDP.rv / ro;
        double w = msh->cells[i]->cellFDP.rw / ro;
        double c_2 =  msh->cells[i]->cellFDP.gamma * msh->cells[i]->cellFDP.P / ro;

	   fprintf(out, "%25.16f\n", sqrt( (u*u + v*v + w*w) / c_2 ) );
	}

    fprintf(out, "VECTORS Velocity double \n");
    for (int i = 0; i < cellCount; i++)
    {
	   double ro = msh->cells[i]->cellFDP.ro;
	   fprintf(out, "%f %f %f\n", msh->cells[i]->cellFDP.ru/ro, msh->cells[i]->cellFDP.rv/ro, msh->cells[i]->cellFDP.rw/ro);
    }

    fclose(out);
}
