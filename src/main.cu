#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "global.h"
#include "solvers/Solver.h"
#include <ctime>
#include <stdio.h>
#include <iostream>




int main(int argc, char** argv)
{
	double te;

	Logger::Instance()->open_log_file("task.log");
    te = clock();

	Method* m = Solver::initMethod("task.xml");
    Solver::runMethod(m);
    Solver::destroyMethod(m);


    cout << endl << "time of execution : " << (clock() - te) / CLOCKS_PER_SEC /60 /60 << " hours"<< endl;
    Logger::Instance()->close_log_file();


    return 0;
}
